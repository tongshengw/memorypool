#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<random>
#include<hip/hip_runtime.h>
#include<memorypool/alloc.h>

// cannot be reorganised
#include<memorypool/math/linalg.h>

unsigned int global_blocksize;

#define MATRIX_CALCULATION_REPEATS 100

void cpu_generate_matrices(double *output, unsigned int number, unsigned int size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(0, 100);
    for(unsigned int i = 0; i < number; i++) {
        for(unsigned int j = 0; j < size * size; j++) {
            output[i * size * size + j] = dis(gen);
        }
    }
}

__global__ void init_all_pools(void *poolMemoryBlock, unsigned int number) {
    unsigned int threadInd = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadInd < number) {
        poolinit(poolMemoryBlock, threadInd);
    }
}

// __global__ void test_ludcmp_kernel(double *matrices, int *results, unsigned int number, unsigned int size, double *buf 
__global__ void test_ludcmp_kernel(double *matrices, int *results, unsigned int number, unsigned int size) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < number) {
        for (unsigned int i = 0; i < MATRIX_CALCULATION_REPEATS; i++) {
            ludcmp(matrices + (index * size * size), results + (index * size), size);
            // ludcmp_buffered(matrices + (index * size * size), results + (index * size), size, buf + (index * size));
        }
    }
}

void test_ludcmp(double *h_input, unsigned int number, unsigned int size) {
    double *d_input;
    hipError_t err = hipMalloc(&d_input, number * size * size * sizeof(double));
    if (err != hipSuccess) {
        printf("Error allocating device memory: %s\n", hipGetErrorString(err));
        return;
    }
    hipMemcpy(d_input, h_input, number * size * size * sizeof(double), hipMemcpyHostToDevice);

    int *d_idx;
    err = hipMalloc(&d_idx, number * size * sizeof(int));
    if (err != hipSuccess) {
        printf("Error allocating device memory for index array: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        return;
    }


    int blockSize = 1024;
    int gridSize = (number + blockSize - 1) / blockSize;

    #ifdef USE_MEMORY_POOL
    void *poolMemoryBlock = allocatePools(number);
    if (poolMemoryBlock == nullptr) {
        printf("Error allocating memory pool\n");
        hipFree(d_input);
        hipFree(d_idx);
        return;
    }
    
    init_all_pools<<<gridSize, blockSize>>>(poolMemoryBlock, number);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("init pools error: %s\n", hipGetErrorString(err));
    }
    #endif
    
    // double *buf;
    // hipMalloc(&buf, number * size * sizeof(double));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // test_ludcmp_kernel<<<gridSize, blockSize>>>(d_input, d_idx, number, size, buf);
    test_ludcmp_kernel<<<gridSize, blockSize>>>(d_input, d_idx, number, size);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel run error: %s\n", hipGetErrorString(err));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f\n", milliseconds);

    hipFree(d_input);
    hipFree(d_idx);
}

__global__ void test_luminv_kernel(double *matrices, double *output, int *idx, unsigned int number, unsigned int size) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < number) {
        for (unsigned int i = 0; i < MATRIX_CALCULATION_REPEATS; i++) {
            ludcmp(matrices + (index * size * size), idx + (index * size), size);
            luminv(output + (index * size * size), matrices + (index * size * size), idx + (index * size), size);
        }
    }
}

void test_luminv(double *h_input, unsigned int number, unsigned int size) {
    double *d_input;
    hipMalloc(&d_input, number * size * size * sizeof(double));
    hipMemcpy(d_input, h_input, number * size * size * sizeof(double), hipMemcpyHostToDevice);

    double *d_output;
    hipMalloc(&d_output, number * size * size * sizeof(double));

    int *d_idx;
    hipMalloc(&d_idx, number * size * sizeof(int));

    int blockSize = 1024;
    int gridSize = (number + blockSize - 1) / blockSize;

    #ifdef USE_MEMORY_POOL
    void *poolMemoryBlock = allocatePools(number);
    init_all_pools<<<gridSize, blockSize>>>(poolMemoryBlock, number);
    hipDeviceSynchronize();
    #endif

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    test_luminv_kernel<<<gridSize, blockSize>>>(d_input, d_output, d_idx, number, size);

    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f\n", milliseconds);

    hipFree(d_input);
    hipFree(d_output);
}

__global__ void test_leastsq_kernel(double *vectors, double *inputs, unsigned int number, unsigned int size) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < number) {
        for (unsigned int i = 0; i < MATRIX_CALCULATION_REPEATS; i++) {
            leastsq(vectors + (index * size), inputs + (index * size * size), size, size);
        }
    }
}

void test_leastsq(double *h_matrices, unsigned int number, unsigned int size) {
    // am lazy, generate size * size inputs, but only size will be used
    double *h_vectors = (double *)malloc(number * size * size * sizeof(double));
    cpu_generate_matrices(h_vectors, number, size);
    // for (unsigned int i = 0; i < number; i++) {
    //     for (unsigned int j = 0; j < size; j++) {
    //         h_vectors[i * size + j] = (double)(j);
    //     }
    // }

    double *d_vectors;
    hipMalloc(&d_vectors, number * size * sizeof(double));
    hipMemcpy(d_vectors, h_vectors, number * size * sizeof(double), hipMemcpyHostToDevice);
    
    double *d_matrices;
    hipMalloc(&d_matrices, number * size * size * sizeof(double));
    hipMemcpy(d_matrices, h_matrices, number * size * size * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 1024;
    int gridSize = (number + blockSize - 1) / blockSize;

    #ifdef USE_MEMORY_POOL
    void *poolMemoryBlock = allocatePools(number);
    init_all_pools<<<gridSize, blockSize>>>(poolMemoryBlock, number);
    hipDeviceSynchronize();
    #endif

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    test_leastsq_kernel<<<gridSize, blockSize>>>(d_vectors, d_matrices, number, size);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f\n", milliseconds);

    hipFree(d_vectors);
    hipFree(d_vectors);
}

int main(int argc, char **argv) {
    if(argc != 4 && argc != 5) {
        printf("Usage: %s <function number> <number of matrices> <size of matrices> optional: <block size>\n", argv[0]);
        printf("Function numbers:\n");
        printf("0: ludcmp\n");
        printf("1: luminv\n");
        printf("2: leastsq\n");
        return 1;
    }

    int function_number = atoi(argv[1]);
    unsigned int number_of_matrices = atoi(argv[2]);
    unsigned int size_of_matrices = atoi(argv[3]);
    if (argc == 5) {
        global_blocksize = atoi(argv[4]);
    } else {
        global_blocksize = 1024;
    }
    // int function_number = 0;
    // unsigned int number_of_matrices = 100000;
    // unsigned int size_of_matrices = 10;

    double *h_input = (double *)malloc(number_of_matrices * size_of_matrices * size_of_matrices * sizeof(double));
    cpu_generate_matrices(h_input, number_of_matrices, size_of_matrices);
    
    // FIXME: comment out to use python script
    // #ifdef USE_MEMORY_POOL
    // printf("Using memory pool\n");
    // #else
    // printf("Not using memory pool\n");
    // #endif
    
    hipDeviceSetLimit(hipLimitMallocHeapSize, 6 * 1024 * 1024 * 1024);

    switch(function_number) {
        case 0:
            test_ludcmp(h_input, number_of_matrices, size_of_matrices);
            break;
        case 1:
            test_luminv(h_input, number_of_matrices, size_of_matrices);
            break;
        case 2:
            test_leastsq(h_input, number_of_matrices, size_of_matrices);
            break;
        default:
            printf("Invalid function number\n");
            free(h_input);
            return 1;
    }

    free(h_input);
    return 0;
}
