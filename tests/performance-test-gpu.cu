#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<random>
#include<hip/hip_runtime.h>
#include<memorypool/alloc.h>
#include<memorypool/math/linalg.h>
#include<memorypool/gpu/poolalloc.cuh>

void cpu_generate_matrices(double *output, unsigned int number, unsigned int size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(0, 100);
    for(unsigned int i = 0; i < number; i++) {
        for(unsigned int j = 0; j < size * size; j++) {
            output[i * size * size + j] = dis(gen);
        }
    }
}

__global__ void init_all_pools(void *poolMemoryBlock, unsigned int number) {
    unsigned int threadInd = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadInd < number) {
        poolinit(poolMemoryBlock, threadInd);
    }
}

// __global__ void test_ludcmp_kernel(double *matrices, int *results, unsigned int number, unsigned int size, double *buf 
__global__ void test_ludcmp_kernel(double *matrices, int *results, unsigned int number, unsigned int size
                                   #ifdef USE_MEMORY_POOL
                                   , void *ptr
                                   #endif
                                   ) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < number) {
        for (unsigned int i = 0; i < 10000; i++) {
            ludcmp(matrices + (index * size * size), results + (index * size), size);
            // ludcmp_buffered(matrices + (index * size * size), results + (index * size), size, buf + (index * size));
        }
    }
}

void test_ludcmp(double *h_input, unsigned int number, unsigned int size) {
    double *d_input;
    hipError_t err = hipMalloc(&d_input, number * size * size * sizeof(double));
    if (err != hipSuccess) {
        printf("Error allocating device memory: %s\n", hipGetErrorString(err));
        return;
    }
    hipMemcpy(d_input, h_input, number * size * size * sizeof(double), hipMemcpyHostToDevice);

    int *d_idx;
    err = hipMalloc(&d_idx, number * size * sizeof(int));
    if (err != hipSuccess) {
        printf("Error allocating device memory for index array: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        return;
    }


    int blockSize = 1024;
    int gridSize = (number + blockSize - 1) / blockSize;

    #ifdef USE_MEMORY_POOL
    void *poolMemoryBlock = allocatePools(number);
    init_all_pools<<<gridSize, blockSize>>>(poolMemoryBlock, number);
    hipDeviceSynchronize();
    #endif
    
    // double *buf;
    // hipMalloc(&buf, number * size * sizeof(double));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    #ifdef USE_MEMORY_POOL
    test_ludcmp_kernel<<<gridSize, blockSize>>>(d_input, d_idx, number, size, poolMemoryBlock);
    #else
    // test_ludcmp_kernel<<<gridSize, blockSize>>>(d_input, d_idx, number, size, buf);
    test_ludcmp_kernel<<<gridSize, blockSize>>>(d_input, d_idx, number, size);
    #endif
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f\n", milliseconds);

    hipFree(d_input);
    hipFree(d_idx);
}

__global__ void test_luminv_kernel(double *matrices, double *output, int *idx, unsigned int number, unsigned int size
                                   #ifdef USE_MEMORY_POOL
                                   , void *ptr
                                   #endif
                                   ) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < number) {
        for (unsigned int i = 0; i < 10000; i++) {
            ludcmp(matrices + (index * size * size), idx + (index * size), size);
            luminv(output + (index * size * size), matrices + (index * size * size), idx + (index * size), size);
        }
    }
}

void test_luminv(double *h_input, unsigned int number, unsigned int size) {
    double *d_input;
    hipMalloc(&d_input, number * size * size * sizeof(double));
    hipMemcpy(d_input, h_input, number * size * size * sizeof(double), hipMemcpyHostToDevice);

    double *d_output;
    hipMalloc(&d_output, number * size * size * sizeof(double));

    int *d_idx;
    hipMalloc(&d_idx, number * size * sizeof(int));

    int blockSize = 1024;
    int gridSize = (number + blockSize - 1) / blockSize;

    #ifdef USE_MEMORY_POOL
    void *poolMemoryBlock = allocatePools(number);
    init_all_pools<<<gridSize, blockSize>>>(poolMemoryBlock, number);
    hipDeviceSynchronize();
    #endif

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    #ifdef USE_MEMORY_POOL
    test_luminv_kernel<<<gridSize, blockSize>>>(d_input, d_output, d_idx, number, size, poolMemoryBlock);
    #else
    test_luminv_kernel<<<gridSize, blockSize>>>(d_input, d_output, d_idx, number, size);
    #endif
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f\n", milliseconds);

    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char **argv) {
    if(argc != 4) {
        printf("Usage: %s <function number> <number of matrices> <size of matrices>\n", argv[0]);
        printf("Function numbers:\n");
        printf("0: ludcmp\n");
        printf("1: luminv\n");
        return 1;
    }

    int function_number = atoi(argv[1]);
    unsigned int number_of_matrices = atoi(argv[2]);
    unsigned int size_of_matrices = atoi(argv[3]);

    double *h_input = (double *)malloc(number_of_matrices * size_of_matrices * size_of_matrices * sizeof(double));
    cpu_generate_matrices(h_input, number_of_matrices, size_of_matrices);
    
    // FIXME: comment out to use python script
    // #ifdef USE_MEMORY_POOL
    // printf("Using memory pool\n");
    // #else
    // printf("Not using memory pool\n");
    // #endif

    switch(function_number) {
        case 0:
            test_ludcmp(h_input, number_of_matrices, size_of_matrices);
            break;
        case 1:
            test_luminv(h_input, number_of_matrices, size_of_matrices);
        default:
            printf("Invalid function number\n");
            free(h_input);
            return 1;
    }

    free(h_input);
    return 0;
}
