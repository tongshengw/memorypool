#include "hip/hip_runtime.h"
#include<stdio.h>
#include<random>
#include<hip/hip_runtime.h>
#include<memorypool/math/linalg.h>

#define SEED 0

void cpu_generate_matrices(double *output, int number, int size) {
    std::mt19937 gen(SEED);
    std::uniform_real_distribution<double> dis(0, 100);
    for(int i = 0; i < number; i++) {
        for(int j = 0; j < size; j++) {
            output[i * size + j] = dis(gen);
        }
    }
}

__global__ void test_ludcmp_kernel(double *matrices, int *results, int number, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < number) {
        ludcmp(matrices + (index * size), results + (index * size), number);
    }
}

void test_ludcmp(double *h_input, int number, int size) {
    double *d_input;
    hipMalloc(&d_input, number * size * sizeof(double));
    hipMemcpy(d_input, h_input, number * size * sizeof(double), hipMemcpyHostToDevice);

    int *d_results;
    hipMalloc(&d_results, number * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    test_ludcmp_kernel<<<number / 1024, 1024>>>(d_input, d_results, number, size);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    int *h_results = (int *)malloc(number * sizeof(int));
    hipMemcpy(h_results, d_results, number * sizeof(int), hipMemcpyDeviceToHost);

    int *ref_results = (int *)malloc(number * sizeof(int));
    for(int i = 0; i < number; i++) {
        ref_results[i] = ludcmp(h_input + (i * size), ref_results + (i * size), size);
        if(h_results[i] != ref_results[i]) {
            printf("Error at index %d: %d != %d\n", i, h_results[i], ref_results[i]);
            exit(1);
        }
    }

    printf("Time taken: %f milliseconds\n", milliseconds);

    hipFree(d_input);
    hipFree(d_results);
    free(h_results);
    free(ref_results);
}

int main(int argc, char **argv) {
    if(argc != 4) {
        printf("Usage: %s <function number> <number of matrices> <size of matrices>\n", argv[0]);
        printf("Function numbers:\n");
        printf("0: ludcmp\n");
        return 1;
    }

    int function_number = atoi(argv[1]);
    int number_of_matrices = atoi(argv[2]);
    int size_of_matrices = atoi(argv[3]);

    double *h_input = (double *)malloc(number_of_matrices * size_of_matrices * sizeof(double));
    cpu_generate_matrices(h_input, number_of_matrices, size_of_matrices);

    switch(function_number) {
        case 0:
            test_ludcmp(h_input, number_of_matrices, size_of_matrices);
            break;
        default:
            printf("Invalid function number\n");
            return 1;
    }
}