#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<random>
#include<hip/hip_runtime.h>
#include<memorypool/math/linalg.h>

// #define SEED 1235
#define APPROX_EQUAL_DIFF 1e-6

bool approx_equal(double a, double b) {
    return fabs(a - b) < APPROX_EQUAL_DIFF;
}

void cpu_generate_matrices(double *output, unsigned int number, unsigned int size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(0, 100);
    for(unsigned int i = 0; i < number; i++) {
        for(unsigned int j = 0; j < size * size; j++) {
            output[i * size * size + j] = dis(gen);
        }
    }
}

__global__ void test_ludcmp_kernel(double *matrices, int *results, unsigned int number, unsigned int size) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < number) {
        ludcmp(matrices + (index * size * size), results + (index * size), size);
    }
}

void test_ludcmp(double *h_input, unsigned int number, unsigned int size) {
    double *d_input;
    hipMalloc(&d_input, number * size * size * sizeof(double));
    hipMemcpy(d_input, h_input, number * size * size * sizeof(double), hipMemcpyHostToDevice);

    int *d_idx;
    hipMalloc(&d_idx, number * size * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Calculate grid size to handle all matrices
    int blockSize = 256;
    int gridSize = (number + blockSize - 1) / blockSize;
    test_ludcmp_kernel<<<gridSize, blockSize>>>(d_input, d_idx, number, size);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // double *gpu_input_modified = (double *)malloc(number * size * size * sizeof(double));
    // hipMemcpy(gpu_input_modified, d_input, number * size * size * sizeof(double), hipMemcpyDeviceToHost);

    // int *gpu_idx_modified = (int *)malloc(number * size * sizeof(int));
    // hipMemcpy(gpu_idx_modified, d_idx, number * size * sizeof(int), hipMemcpyDeviceToHost);

    // int *ref_idx = (int *)malloc(number * size * sizeof(int));

    // // printf("Calculating CPU reference...\n");
    // for (unsigned int i = 0; i < number; i++) {
    //     ludcmp(h_input + (i * size * size), ref_idx + (i * size), size);
    // }

    // // checking idx array
    // // printf("Checking idx array correctness...\n");
    // for (unsigned int i = 0; i < number; i++) {
    //     for (unsigned int j = 0; j < size; j++) {
    //         int idx_pos = i * size + j;
    //         if(gpu_idx_modified[idx_pos] != ref_idx[idx_pos]) {
    //             printf("Error at idx index %d (matrix %d, element %d): %d != %d\n", idx_pos, i, j, gpu_idx_modified[idx_pos], ref_idx[idx_pos]);
    //             exit(1);
    //         }
    //     }
    // }
    
    // // checking input array
    // // printf("Checking input array correctness...\n");
    // for (unsigned int i = 0; i < number; i++) {
    //     for (unsigned int j = 0; j < size * size; j++) {
    //         if(!approx_equal(h_input[i * size * size + j], gpu_input_modified[i * size * size + j])) {
    //             printf("Error at input index %d: %f != %f\n", i * size * size + j, gpu_input_modified[i * size * size + j], h_input[i * size * size + j]);
    //             exit(1);
    //         }
    //     }
    // }

    printf("%f\n", milliseconds);

    // hipFree(d_input);
    // hipFree(d_idx);
    // free(gpu_input_modified);
    // free(gpu_idx_modified);
    // free(ref_idx);
}

int main(int argc, char **argv) {
    if(argc != 4) {
        printf("Usage: %s <function number> <number of matrices> <size of matrices>\n", argv[0]);
        printf("Function numbers:\n");
        printf("0: ludcmp\n");
        return 1;
    }

    int function_number = atoi(argv[1]);
    unsigned int number_of_matrices = atoi(argv[2]);
    unsigned int size_of_matrices = atoi(argv[3]);

    double *h_input = (double *)malloc(number_of_matrices * size_of_matrices * size_of_matrices * sizeof(double));
    cpu_generate_matrices(h_input, number_of_matrices, size_of_matrices);
    // printf("h_input values:\n");
    // for (int i = 0; i < number_of_matrices; i++) {
    //     printf("Matrix %d:\n", i);
    //     for (int j = 0; j < size_of_matrices; j++) {
    //         for (int k = 0; k < size_of_matrices; k++) {
    //             printf("%f ", h_input[i * size_of_matrices * size_of_matrices + j * size_of_matrices + k]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }

    switch(function_number) {
        case 0:
            test_ludcmp(h_input, number_of_matrices, size_of_matrices);
            break;
        default:
            printf("Invalid function number\n");
            free(h_input);
            return 1;
    }

    free(h_input);
    return 0;
}