#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "linalg.h"

__global__ void test_linalg_kernel() {
    test_ludcmp();
    test_lubksb();
    test_luminv();
    test_leastsq();
    test_leastsq_kkt();
}

void run_linalg_tests() {
    test_linalg_kernel<<<1, 256>>>();
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        exit(1);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error after kernel execution: %s\n", hipGetErrorString(err));
        exit(1);
    }
}
