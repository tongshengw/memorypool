#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "poolalloc.cuh"

__device__ int pool_lock = 0;

__device__ void lock(int *mutex) {
    while (atomicCAS(mutex, 0, 1) != 0) {
        // spin
    }
}

__device__ void unlock(int *mutex) {
    atomicExch(mutex, 0);
}


__global__ void allocate_and_write(int **ptrs, int n, void *poolMemoryBlock) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    lock(&pool_lock);
    poolinit(poolMemoryBlock, idx);
    unlock(&pool_lock);

    if (idx < n) {

        lock(&pool_lock);
        int *mem = (int*)poolmalloc(4 * sizeof(int));
        unlock(&pool_lock);


        if (mem != NULL) {
            for (int i = 0; i < 4; ++i) {
                mem[i] = idx * 10 + i;
            }
            ptrs[idx] = mem;
        } else {
            ptrs[idx] = NULL;
        }
    }
}

__global__ void read_and_free(int **ptrs, int n) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Thread %d: ", idx);
    if (idx < n && ptrs[idx] != NULL) {
        for (int i = 0; i < 4; ++i) {
            printf("%d ", ptrs[idx][i]);
        }
        poolfree(ptrs[idx]);
    }
}

int main() {
    int n = 8;
    int **d_ptrs;
    printf("here");
    hipMalloc(&d_ptrs, n * sizeof(int*));

    void *poolPtr = allocatePools(n);
    allocate_and_write<<<1, n>>>(d_ptrs, n, poolPtr);
    hipDeviceSynchronize();

    read_and_free<<<1, n>>>(d_ptrs, n);
    hipDeviceSynchronize();
    freePools(poolPtr);

    hipFree(d_ptrs);
    return 0;
}
