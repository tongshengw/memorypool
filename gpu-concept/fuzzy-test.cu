#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unordered_set>
#include <getopt.h>
#include <assert.h>

#include "poolalloc.cuh"

#define NUM_THREADS 4
#define OPS_PER_THREAD 10

struct TestOperation {
    bool isAlloc;
    unsigned long numBytes;
    int corrospondingAlloc;
};

void generateRandomOperations(TestOperation *ops) {
    for (int i = 0; i < NUM_THREADS; i++) {
        std::unordered_set<size_t> allocationIndices;
        size_t startingInd = i * OPS_PER_THREAD;
        for (int j = 0; j < OPS_PER_THREAD; j++) {
            if (allocationIndices.size() == 0) {
                ops[startingInd + j].isAlloc = true;
                ops[startingInd + j].numBytes = 4 * sizeof(int);
                ops[startingInd + j].corrospondingAlloc = -1;
                allocationIndices.insert(j);
            } else if (rand() % 2 == 0) {
                ops[startingInd + j].isAlloc = true;
                ops[startingInd + j].numBytes = 4 * sizeof(int);
                ops[startingInd + j].corrospondingAlloc = -1;
                allocationIndices.insert(startingInd + j);
            } else {
                ops[startingInd + j].isAlloc = false;
                ops[startingInd + j].numBytes = 0;
                auto it = allocationIndices.begin();
                std::advance(it, rand() % allocationIndices.size());
                ops[startingInd + j].corrospondingAlloc = *it;
                allocationIndices.erase(it);
            }
        }
    }
}

// Simple global spinlock for serialization
__device__ int global_lock = 0;

__device__ void acquire_lock(int *lock) {
    while (atomicCAS(lock, 0, 1) != 0) {
        // spin
    }
    __threadfence();
}

__device__ void release_lock(int *lock) {
    __threadfence();
    atomicExch(lock, 0);
}

__global__ void runTests(TestOperation *ops, void *poolMemoryBlock) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    poolinit(poolMemoryBlock, idx);

    void *allocatedPtrs[OPS_PER_THREAD];

    // acquire_lock(&global_lock);

    for (unsigned int i = 0; i < OPS_PER_THREAD; i++) {
        unsigned int opIndex = idx * OPS_PER_THREAD + i;
        if (ops[opIndex].isAlloc) {
            void *ptr = poolmalloc(ops[opIndex].numBytes);
            allocatedPtrs[i] = ptr;
            for (unsigned long j = 0; j < ops[opIndex].numBytes; j++) {
                ((char*)ptr)[j] = 'a';
            }
            printf("Thread %d: Allocated %lu bytes at %p\n", idx, ops[opIndex].numBytes, ptr);
        } else {
            char *ptrToFree = (char*)allocatedPtrs[ops[opIndex].corrospondingAlloc];
            for (unsigned long j = 0; j < ops[ops[opIndex].corrospondingAlloc].numBytes; j++) {
                if (ptrToFree[j] != 'a') {
                    printf("Thread %d: failed to free allocation at index %d\n", idx, ops[opIndex].corrospondingAlloc);
                    assert(ptrToFree[j] == 'a');
                }
            }
            poolfree(ptrToFree);
            printf("Thread %d: Freed allocation at index %d\n", idx, ops[opIndex].corrospondingAlloc);
        }
    }

    // release_lock(&global_lock);
}

int main(int argc, char **argv) {
    int opt;
    int seed = 0;
    while ((opt = getopt(argc, argv, "s:")) != -1) {
        switch (opt) {
            case 's':
                seed = atoi(optarg);
                break;
            default:
                fprintf(stderr, "Usage: %s [-s seed]\n", argv[0]);
                exit(1);
        }
    }
    srand(seed);
    printf("Random seed: %d\n", seed);

    TestOperation ops[NUM_THREADS * OPS_PER_THREAD];
    generateRandomOperations(ops);
    
    TestOperation *d_ops;
    hipMalloc(&d_ops, sizeof(TestOperation) * NUM_THREADS * OPS_PER_THREAD);
    hipMemcpy(d_ops, ops, sizeof(TestOperation) * NUM_THREADS * OPS_PER_THREAD, hipMemcpyHostToDevice);

    runTests<<<1, NUM_THREADS>>>(d_ops, allocatePools(NUM_THREADS));
    hipDeviceSynchronize();
    hipFree(d_ops);
}